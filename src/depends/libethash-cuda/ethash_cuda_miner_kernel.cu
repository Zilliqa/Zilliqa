#include "hip/hip_runtime.h"
/*
* Genoil's CUDA mining kernel for Ethereum
* based on Tim Hughes' opencl kernel.
* thanks to sp_, trpuvot, djm34, cbuchner for things i took from ccminer.
*/
#include "ethash_cuda_miner_kernel.h"
#include "ethash_cuda_miner_kernel_globals.h"
#include "cuda_helper.h"

#include "fnv.cuh"

#define copy(dst, src, count) for (int i = 0; i != count; ++i) { (dst)[i] = (src)[i]; }

#include "keccak.cuh"
#include "dagger_shuffled.cuh"

template <uint32_t _PARALLEL_HASH>
__global__ void 
ethash_search(
	volatile search_results* g_output,
	uint64_t start_nonce
	)
{
	uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
	uint2 mix[4];
        if (compute_hash<_PARALLEL_HASH>(start_nonce + gid, d_target, mix))
		return;
	uint32_t index = atomicInc((uint32_t *)&g_output->count, 0xffffffff);
	if (index >= SEARCH_RESULTS)
		return;
	g_output->result[index].gid = gid;
	g_output->result[index].mix[0] = mix[0].x;
	g_output->result[index].mix[1] = mix[0].y;
	g_output->result[index].mix[2] = mix[1].x;
	g_output->result[index].mix[3] = mix[1].y;
	g_output->result[index].mix[4] = mix[2].x;
	g_output->result[index].mix[5] = mix[2].y;
	g_output->result[index].mix[6] = mix[3].x;
	g_output->result[index].mix[7] = mix[3].y;
}

void run_ethash_search(
	uint32_t gridSize,
	uint32_t blockSize,
	hipStream_t stream,
	volatile search_results* g_output,
	uint64_t start_nonce,
	uint32_t parallelHash
)
{
	switch (parallelHash)
	{
		case 1: ethash_search <1> <<<gridSize, blockSize, 0, stream >>>(g_output, start_nonce); break;
		case 2: ethash_search <2> <<<gridSize, blockSize, 0, stream >>>(g_output, start_nonce); break;
		case 4: ethash_search <4> <<<gridSize, blockSize, 0, stream >>>(g_output, start_nonce); break;
		case 8: ethash_search <8> <<<gridSize, blockSize, 0, stream >>>(g_output, start_nonce); break;
		default: ethash_search <4> <<<gridSize, blockSize, 0, stream >>>(g_output, start_nonce); break;
	}
	CUDA_SAFE_CALL(hipGetLastError());
}

#define ETHASH_DATASET_PARENTS 256
#define NODE_WORDS (64/4)


__global__ void
ethash_calculate_dag_item(uint32_t start)
{
	uint32_t const node_index = start + blockIdx.x * blockDim.x + threadIdx.x;
	if (((node_index/4)*4) >= d_dag_size * 2) return;

	hash200_t dag_node;
	copy(dag_node.uint4s, d_light[node_index % d_light_size].uint4s, 4);
	dag_node.words[0] ^= node_index;
	SHA3_512(dag_node.uint2s);

	const int thread_id = threadIdx.x & 3;

	for (uint32_t i = 0; i != ETHASH_DATASET_PARENTS; ++i) {
		uint32_t parent_index = fnv(node_index ^ i, dag_node.words[i % NODE_WORDS]) % d_light_size;
		for (uint32_t t = 0; t < 4; t++) {

			uint32_t shuffle_index = __shfl_sync(0xFFFFFFFF,parent_index, t, 4);

			uint4 p4 = d_light[shuffle_index].uint4s[thread_id];
			for (int w = 0; w < 4; w++) {

				uint4 s4 = make_uint4(__shfl_sync(0xFFFFFFFF,p4.x, w, 4), __shfl_sync(0xFFFFFFFF,p4.y, w, 4), __shfl_sync(0xFFFFFFFF,p4.z, w, 4), __shfl_sync(0xFFFFFFFF,p4.w, w, 4));
				if (t == thread_id) {
					dag_node.uint4s[w] = fnv4(dag_node.uint4s[w], s4);
				}
			}
		}
	}
	SHA3_512(dag_node.uint2s);
	hash64_t * dag_nodes = (hash64_t *)d_dag;

	for (uint32_t t = 0; t < 4; t++) {
		uint32_t shuffle_index = __shfl_sync(0xFFFFFFFF,node_index, t, 4);
		uint4 s[4];
		for (uint32_t w = 0; w < 4; w++) {
			s[w] = make_uint4(__shfl_sync(0xFFFFFFFF,dag_node.uint4s[w].x, t, 4), __shfl_sync(0xFFFFFFFF,dag_node.uint4s[w].y, t, 4), __shfl_sync(0xFFFFFFFF,dag_node.uint4s[w].z, t, 4), __shfl_sync(0xFFFFFFFF,dag_node.uint4s[w].w, t, 4));
		}
		if (shuffle_index < d_dag_size * 2) {
		dag_nodes[shuffle_index].uint4s[thread_id] = s[thread_id];
	}
}
}

void ethash_generate_dag(
	uint64_t dag_size,
	uint32_t blocks,
	uint32_t threads,
	hipStream_t stream
	)
{
	const uint32_t work = (uint32_t)(dag_size / sizeof(hash64_t));
	const uint32_t run = blocks * threads;

	for (uint32_t base = 0; base < work; base += run)
	{
		ethash_calculate_dag_item <<<blocks, threads, 0, stream>>>(base);
		CUDA_SAFE_CALL(hipDeviceSynchronize());
	}
	CUDA_SAFE_CALL(hipGetLastError());
}

void set_constants(
	hash128_t* _dag,
	uint32_t _dag_size,
	hash64_t * _light,
	uint32_t _light_size
	)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_dag), &_dag, sizeof(hash128_t *)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), &_dag_size, sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_light), &_light, sizeof(hash64_t *)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_light_size), &_light_size, sizeof(uint32_t)));
}

void set_header(
	hash32_t _header
	)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_header), &_header, sizeof(hash32_t)));
}

void set_target(
	uint64_t _target
	)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_target), &_target, sizeof(uint64_t)));
}
